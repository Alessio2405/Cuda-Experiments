#include <ctime>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>


// CUDA Kernel function to compute the dot product of two vectors
__global__ void computeDotProduct(const float *vectorFirst, const float *vectorSecond, float *dotProductResult, int vectorSize) {
    int threadIndex = blockDim.x * blockIdx.x + threadIdx.x;
    if (threadIndex < vectorSize) {
        atomicAdd(dotProductResult, vectorFirst[threadIndex] * vectorSecond[threadIndex]);
    }
}

// Function to generate a random float
float generateRandomFloat(int maxValue = 1000) {
    return static_cast<float>(rand()) / static_cast<float>(maxValue);
}

int main() {
    srand(time(0));

    int vectorSize = 300;
    size_t memorySize = vectorSize * sizeof(float);

    float *hostVectorA, *hostVectorB, *hostResult;
    float *deviceVectorA, *deviceVectorB, *deviceResult;

    hostVectorA = (float*) malloc(memorySize);
    hostVectorB = (float*) malloc(memorySize);
    hostResult = (float*) malloc(sizeof(float));

    for (int index = 0; index < vectorSize; index++) {
        hostVectorA[index] = generateRandomFloat();
        hostVectorB[index] = generateRandomFloat();
    }

    hipMalloc((void**)&deviceVectorA, memorySize);
    hipMalloc((void**)&deviceVectorB, memorySize);
    hipMalloc((void**)&deviceResult, sizeof(float));

    hipMemcpy(deviceVectorA, hostVectorA, vectorSize, hipMemcpyHostToDevice);
    hipMemcpy(deviceVectorB, hostVectorB, vectorSize, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (vectorSize + threadsPerBlock - 1) / threadsPerBlock;

    computeDotProduct<<<blocksPerGrid, threadsPerBlock>>>(deviceVectorA, deviceVectorB, deviceResult, vectorSize);

    hipMemcpy(hostResult, deviceResult, sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Dot product: " << *hostResult << std::endl;

    free(hostVectorA);
    free(hostVectorB);
    free(hostResult);

    hipFree(deviceVectorA);
    hipFree(deviceVectorB);
    hipFree(deviceResult);
}
