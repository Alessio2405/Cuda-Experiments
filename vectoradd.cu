#include <iostream> // for std::cout
#include <hip/hip_runtime.h> // for CUDA runtime functions
 // for CUDA kernel launch parameters


__global__ void vectorAdd(const int *a, 
                          const int *b, 
                          int *c, 
                          int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements){
        c[i] = a[i] + b[i];
    }
}

int main() {
    int numElements = 50;

    size_t size = numElements * sizeof(int);
    int *hostA, *hostB, *hostC;
    int *deviceA, *deviceB, *deviceC;
    hostA = (int*) malloc(size);
    hostB = (int*) malloc(size);
    hostC = (int*) malloc(size);

    for (int i = 0; i < numElements; i++){
        hostA[i] = i;
        hostB[i] = i*2;
    }

    hipMalloc((void**)&deviceA, size);
    hipMalloc((void**)&deviceB, size);
    hipMalloc((void**)&deviceC, size);

    hipMemcpy(deviceA, hostA, size, hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(deviceA, deviceB, deviceC, numElements);

    hipMemcpy(hostC, deviceC, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < numElements; i++){
        std::cout << hostA[i] << " + " << hostB[i] << " = " << hostC[i] << std::endl;
    }

    free(hostA);
    free(hostB);
    free(hostC);

    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    return 0;
}